#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include "../common/common.hpp"
#include "../common/solver.hpp"
int nx, ny;
double *d_h, *d_u, *d_v;  
double *d_dh, *d_du, *d_dv;  
double *d_dh1, *d_du1, *d_dv1;  
double *d_dh2, *d_du2, *d_dv2;  
int t=0;
double H, g, dx, dy, dt;
#define BLOCK_SIZE 256 
/**
 * This is your initialization function! We pass in h0, u0, and v0, which are
 * your initial height, u velocity, and v velocity fields. You should send these
 * grids to the GPU so you can do work on them there, and also these other fields.
 * Here, length and width are the length and width of the domain, and nx and ny are
 * the number of grid points in the x and y directions. H is the height of the water
 * column, g is the acceleration due to gravity, and dt is the time step size.
 * The rank and num_procs variables are unused here, but you will need them
 * when doing the MPI version.
 */
void init(double *h0, double *u0, double *v0, double length_, double width_, 
          int nx_, int ny_, double H_, double g_, double dt_,
          int rank_, int num_procs_)
{
    // @TODO: your code here
    int size;
    nx = nx_;
    ny = ny_;
    size=nx*ny*sizeof(double);
    hipMalloc(&d_h, (nx + 1) * (ny + 1) * sizeof(double));
    hipMalloc(&d_u, (nx + 1) * ny * sizeof(double));
    hipMalloc(&d_v, nx * (ny + 1) * sizeof(double));
    hipMalloc(&d_dh,size);
    hipMalloc(&d_du,size);
    hipMalloc(&d_dv,size);
    hipMalloc(&d_dh1,size);
    hipMalloc(&d_du1,size);
    hipMalloc(&d_dv1,size);
    hipMalloc(&d_dh2,size);
    hipMalloc(&d_du2,size);
    hipMalloc(&d_dv2,size);

    hipMemcpy(d_h, h0, (nx + 1) * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_u, u0, (nx + 1) * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v0, nx * (ny + 1) * sizeof(double), hipMemcpyHostToDevice);

    H = H_;
    g = g_;

    dx = length_ / nx;
    dy = width_ / ny;

    dt = dt_;

}

__global__ void compute_and_multistep_kernel(double *h, double *u, double *v,
                                    double *dh, double *dh1, double *dh2,
                                    double *du, double *du1, double *du2,
                                    double *dv, double *dv1, double *dv2,
                                    int nx, int ny, double H, double g,
                                    double dx, double dy, double dt,
                                    double a1, double a2, double a3)
{
    __shared__ double s_h[BLOCK_SIZE];
    __shared__ double s_u[BLOCK_SIZE];
    __shared__ double s_v[BLOCK_SIZE];

    int tid = threadIdx.x;
    int global_idx =blockIdx.x*blockDim.x+threadIdx.x;
    int i = global_idx / ny;
    int j = global_idx % ny;

    if(i < nx && j < ny) {
        s_h[tid] = h(i, j);
        s_u[tid] = u(i, j);
        s_v[tid] = v(i, j);

        __syncthreads();

        double local_du_dx = (u(i + 1, j) - s_u[tid]) / dx;
        double local_dv_dy = (v(i, j + 1) - s_v[tid]) / dy;
        double local_dh_dx = (h(i + 1, j) - s_h[tid]) / dx;
        double local_dh_dy = (h(i, j + 1) - s_h[tid]) / dy;

        double current_dh = -H * (local_du_dx + local_dv_dy);
        double current_du = -g * local_dh_dx;
        double current_dv = -g * local_dh_dy;

        h(i, j) += (a1 * current_dh + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
        u(i + 1, j) += (a1 * current_du + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
        v(i, j + 1) += (a1 * current_dv + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;

        __syncthreads();
        
        dh(i, j) = current_dh;
        du(i, j) = current_du;
        dv(i, j) = current_dv;
        __syncthreads();
    }
}

/**
 * This function computes the ghost cells for the horizontal boundaries.
 * This is done by copying the values from the opposite side of the domain.
 */
__global__ void compute_boundaries_kernel(double *h, double *u, double *v,
                                       int nx, int ny)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int j = tid; j < ny; j += stride) {
        h(nx, j) = h(0, j);
        u(0, j) = u(nx, j);
    }

    __syncthreads();

    for (int i = tid; i < nx; i += stride) {
        h(i, ny) = h(i, 0);
        v(i, 0) = v(i, ny);
    }
}



/**
 * This is your step function! Here, you will actually numerically solve the shallow
 * water equations. You should update the h, u, and v fields to be the solution after
 * one time step has passed.
 */
void step()
{
    int total_threads = nx * ny;
    int num_blocks = (total_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int boundary_blocks = (max(nx, ny) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    compute_boundaries_kernel<<<boundary_blocks, BLOCK_SIZE>>>(d_h, d_u, d_v, nx, ny);

    double a1, a2 = 0.0, a3 = 0.0;
    if (t == 0) {
        a1 = 1.0;
    } else if (t == 1) {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    } else {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }

    compute_and_multistep_kernel<<<num_blocks, BLOCK_SIZE>>>(d_h, d_u, d_v,
                                                    d_dh, d_dh1, d_dh2,
                                                    d_du, d_du1, d_du2,
                                                    d_dv, d_dv1, d_dv2,
                                                    nx, ny, H, g, dx, dy, dt,
                                                    a1, a2, a3);

    double *tmp;
    tmp = d_dh2; d_dh2 = d_dh1; d_dh1 = d_dh; d_dh = tmp;
    tmp = d_du2; d_du2 = d_du1; d_du1 = d_du; d_du = tmp;
    tmp = d_dv2; d_dv2 = d_dv1; d_dv1 = d_dv; d_dv = tmp;
    
    t++;
}

/**
 * This is your transfer function! You should copy the h field back to the host
 * so that the CPU can check the results of your computation.
 */
void transfer(double *h)
{
    // @TODO: Your code here
    hipMemcpy(h, d_h, nx * ny * sizeof(double),
                               hipMemcpyDeviceToHost);
}

/**
 * This is your finalization function! You should free all of the memory that you
 * allocated on the GPU here.
 */
void free_memory()
{
    // @TODO: Your code here
    hipFree(d_h);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_dh);
    hipFree(d_du);
    hipFree(d_dv);
    hipFree(d_dh1);
    hipFree(d_du1);
    hipFree(d_dv1);
    hipFree(d_dh2);
    hipFree(d_du2);
    hipFree(d_dv2);
}  